#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"

#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <iomanip>
#include <random>
#include <chrono>
#include <fstream>

#include <mutex>
#include <thread>



constexpr int matrixStartIndexRow(const int matrix_dim, const int block_dim, int block_Idx, int block_Idy, int thread_idx) {
    return block_Idx * block_dim * matrix_dim + thread_idx * matrix_dim + block_Idy * block_dim;
}

constexpr int matrixStartIndexColumn(const int matrix_dim, const int block_dim, int block_Idx, int block_Idy, int thread_idx) {
    return block_Idx * block_dim * matrix_dim + block_Idy * block_dim + thread_idx;
}

constexpr int shiftedVectorIndex(const int matrix_dim, const int shift, const int index) {
    return ((shift + index) % matrix_dim) * 2;
}

__global__ void updateWithRowShift(hipFloatComplex* dev_matrix, hipTextureObject_t dev_vector, const int matrix_dim, const int shift)
{

    if (blockIdx.x * blockDim.x + threadIdx.x < matrix_dim) {
        unsigned int start_index = matrixStartIndexColumn(matrix_dim, blockDim.x, blockIdx.x, blockIdx.y, threadIdx.x);
        int end = std::min(matrix_dim - blockIdx.x * blockDim.x, blockDim.x);
        for (int i = 0; i < end; i++) {
            int vectorPos = (shift * (blockIdx.x * blockDim.x + i) + (threadIdx.x + blockIdx.y * blockDim.x)) % matrix_dim * 2;
            dev_matrix[start_index + i * matrix_dim].x += tex2D<float>(dev_vector, vectorPos, 0);
            dev_matrix[start_index + i * matrix_dim].y += tex2D<float>(dev_vector, vectorPos + 1, 0);
        }
    }

}

__global__ void updateWithColumnShift(hipFloatComplex* dev_matrix, hipTextureObject_t dev_vector, const int matrix_dim, const int shift)
{
    if (blockIdx.x * blockDim.x + threadIdx.x < matrix_dim) {
        unsigned int start_index = matrixStartIndexColumn(matrix_dim, blockDim.x, blockIdx.x, blockIdx.y, threadIdx.x);
        int end = std::min(matrix_dim - blockIdx.x * blockDim.x, blockDim.x);
        for (int i = 0; i < end; i++) {
            int vectorPos = shiftedVectorIndex(matrix_dim, shift * threadIdx.x + blockDim.x * blockIdx.y * shift + blockIdx.x * blockDim.x, i);

            dev_matrix[start_index + i * matrix_dim].x += tex2D<float>(dev_vector, vectorPos, 0);
            dev_matrix[start_index + i * matrix_dim].y += tex2D<float>(dev_vector, vectorPos + 1, 0);

        }
    }

}



class spift
{
public:
    spift(const int matrixDim, const int blockDim, const int iterations, const int GPU_index, const int wait);
    ~spift();
    spift(const spift&) = delete;
    hipError_t prepareGPU();
    hipError_t initTexture();
    hipError_t iterate();
    hipError_t iteration(int shift);
    auto displayTime();
    void printResult();
    void generateShifts();
    void initResult();
    void initCoalescence();
    
    void getShiftVector(int i);
    void initShiftTest();
    
    

private:
    //the dimensions of the grid and blocks
    const int matrixDim;
    const int blockDim;

    dim3* blockDim3;
    dim3* gridDim3;

    //the Matrix where the result is loaded into in the end
    hipFloatComplex* result;

    //the device-matrix, where the current state is saved
    hipFloatComplex* dev_matrix;

    //a set of randomgenerated shifts. testing only
    float** shift;

    //the texture object where the current shift is saved during kernel execution
    hipTextureObject_t* texObj = new hipTextureObject_t();

    //the data in texobj
    hipArray* *cuArray = new hipArray * ();

    //the number of iterations, testing only
    int iterations;

    //measuring the execution time
    long long duration;

    //the aggregation of shifts, first half are rowShifts
    float** coalescence;

    //the index, where it is saved, wheter data is aggregated for this shift
    int* coalescenceSet;

    std::mutex** shiftIndexMutex;

    //boolean wheter execution is done
    int* done;

    int wait;

    const int GPUIndex;



};

spift::spift(const int matrixDim, const int blockDim, const int iterations, const int GPU_index, const int wait) : matrixDim(matrixDim), blockDim(blockDim), iterations(iterations), GPUIndex(GPU_index), wait(wait)
{
    hipError_t cudaStatus;

    this->initResult();

    this->initCoalescence();

    this->initShiftTest();


    int nr_blocks = ceil((double)matrixDim / (double)this->blockDim);
    this->blockDim3 = new dim3(this->blockDim, 1);;
    this->gridDim3 = new dim3(nr_blocks, nr_blocks);;

    cudaStatus = prepareGPU();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GPU init failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipMallocArray(cuArray, &channelDesc, matrixDim * 2, 1);

    cudaStatus = this->initTexture();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "GPU texture init failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

}

void spift::initResult() {
    this->result = new hipFloatComplex[this->matrixDim * this->matrixDim];
    //allocate memory for result
    for (int x = 0; x < this->matrixDim * this->matrixDim; ++x) {
        hipFloatComplex next;
        next.x = 0;
        next.y = 0;
        this->result[x] = next;
    }
}

void spift::initCoalescence()
{
    this->shiftIndexMutex = new std::mutex*[matrixDim * 2];
    this->done = new int(0);
    this->coalescenceSet = new int[matrixDim * 2];
    this->coalescence = new float* [matrixDim * 2];

    for (int i = 0; i < matrixDim * 2; ++i) {
        this->coalescenceSet[i] = 0;
        this->shiftIndexMutex[i] = new std::mutex();
        this->coalescence[i] = new float[matrixDim * 2];
    }
}

void spift::initShiftTest() {
    //allocate memory for shift vectors. testing only
    std::uniform_real_distribution<> dist(-1, 1);
    std::random_device rd;
    std::mt19937 gen(rd());
    this->shift = new float* [200];
    for (int i = 0; i < 200; ++i) {
        float* bla = new float[2 * matrixDim];
        for (int x = 0; x < matrixDim * 2; ++x) {
            //bla[x] = x;
            bla[x] = dist(gen);
        }
        this->shift[i] = bla;
    }
}

hipError_t spift::prepareGPU() {
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(this->GPUIndex);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }
    // Allocate GPU buffers for matrix
    cudaStatus = hipSetDevice(this->GPUIndex);
    cudaStatus = hipMalloc((void**)&(this->dev_matrix), this->matrixDim * this->matrixDim * sizeof(hipFloatComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    // Copy input Matrix from host memory to GPU buffers.
    cudaStatus = hipSetDevice(this->GPUIndex);
    cudaStatus = hipMemcpy((this->dev_matrix), (this->result), this->matrixDim * this->matrixDim * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return cudaStatus;
    }
    return hipSuccess;
}

auto spift::displayTime() {
    std::cout << "terminated " << iterations << " iterations,  " << matrixDim << " dim, time: " << duration << "ms waitTime: " << this->wait << " micro seconds" << std::endl;
    std::cout << "time per iteration: " << duration / (double)iterations << std::endl;
    return duration;
}

void spift::printResult() {
    for (int i = 0; i < matrixDim; ++i) {
        for (int j = 0; j < matrixDim; ++j) {
            std::cout << "(" << std::setfill(' ') << std::setw(2) << this->result[i * matrixDim + j].x << ", " << std::setfill(' ') << std::setw(2) << this->result[i * matrixDim + j].y << "), ";
        }
        std::cout << std::endl;
    }
}

void spift::generateShifts() {
    int updates = 0;
    std::uniform_real_distribution<> dist(0, 1);
    std::random_device rd;
    std::mt19937 gen(rd());
    for (int i = 0; i < iterations; ++i) {
        int posiCoal = (int)(dist(gen) * this->matrixDim * 2);
        int posiShift = (int)(dist(gen) * 200);
        this->shiftIndexMutex[posiCoal]->lock();
        if (this->coalescenceSet[posiCoal]) {
            ++updates;
            //std::cout << "updating: " << posiCoal << std::endl;
            for (int j = 0; j < this->matrixDim * 2; ++j) {
                this->coalescence[posiCoal][j] += this->shift[posiShift][j];
            }
        }
        else {
            //std::cout << "first:    " << posiCoal;
            for (int j = 0; j < this->matrixDim * 2; ++j) {
                this->coalescence[posiCoal][j] = this->shift[posiShift][j];
            }
            this->coalescenceSet[posiCoal] = 1;
        }
        std::this_thread::sleep_for(std::chrono::microseconds(this->wait));
        //std::cout << "\t-unlocking" << std::endl;

        this->shiftIndexMutex[posiCoal]->unlock();

    }
    *(this->done) = 1;
}

hipError_t spift::iterate() {
    /*
    Locks needed for:
    - this->coalescenceSet
    - this->coalescence
    */

    hipError_t cudaStatus;
    auto t1 = std::chrono::high_resolution_clock::now();
    while(!*done) {
        //std::cout << "done: " << *done << std::endl;
        for (int shiftPos = 0; shiftPos < this->matrixDim * 2; ++shiftPos) {
            if (this->coalescenceSet[shiftPos]) {
                if (this->shiftIndexMutex[shiftPos]->try_lock()) {
                    this->coalescenceSet[shiftPos] = 0;
                    cudaStatus = iteration(shiftPos);
                    // Check for any errors in iteration
                    if (cudaStatus != hipSuccess) {
                        fprintf(stderr, "iteration %d failed: %s\n", shiftPos, hipGetErrorString(cudaStatus));
                        return cudaStatus;
                    }
                }
            }
        }
    }
    for (int shiftPos = 0; shiftPos < this->matrixDim * 2; ++shiftPos) {
        if (this->coalescenceSet[shiftPos]) {
            if (this->shiftIndexMutex[shiftPos]->try_lock()) {
                this->coalescenceSet[shiftPos] = 0;
                cudaStatus = iteration(shiftPos);
                // Check for any errors in iteration
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "iteration %d failed: %s\n", shiftPos, hipGetErrorString(cudaStatus));
                    return cudaStatus;
                }
            }
        }
    }

    auto t2 = std::chrono::high_resolution_clock::now();
    this->duration = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
    

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(result, dev_matrix, matrixDim * matrixDim * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return cudaStatus;
    }
    return hipSuccess;
}

void spift::getShiftVector(int i) {
    hipSetDevice(this->GPUIndex);
    auto cudaStatus = hipMemcpyToArray(*cuArray, 0, 0, this->coalescence[i], matrixDim * 2 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "updateWithRowShift launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

}

hipError_t spift::iteration(int shift) {
    getShiftVector(shift);
    hipSetDevice(this->GPUIndex);
    if (shift < this->matrixDim / 2) {
        updateWithRowShift << <*gridDim3, *blockDim3 >> > (dev_matrix, *texObj, this->matrixDim, shift);
    }
    else {
        updateWithColumnShift << <*gridDim3, *blockDim3 >> > (dev_matrix, *texObj, this->matrixDim, shift - this->matrixDim);
    }

    // Check for any errors launching the kernel
    auto cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "updateWithRowShift launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipSetDevice(this->GPUIndex);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d: %s after launching Kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    this->shiftIndexMutex[shift]->unlock();
    return hipSuccess;
}


hipError_t spift::initTexture() {


    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = *(this->cuArray);
    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    hipSetDevice(this->GPUIndex);
    hipCreateTextureObject(texObj, &resDesc, &texDesc, NULL);
    auto cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "textureInit failed: %s\n", hipGetErrorString(cudaStatus));
    }
    return hipSuccess;
}

spift::~spift()
{
    free(this->shiftIndexMutex);
    free(this->result);
    for (int i = 0; i < 200; ++i) {
        free(this->shift[i]);
    }
   
    free(this->shift);
    hipSetDevice(this->GPUIndex);
    hipFree(this->dev_matrix);
    hipFreeArray(*(this->cuArray));
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
    for (int i = 0; i < matrixDim * 2; ++i) {
        free(this->coalescence[i]);
    }
    free(this->coalescenceSet);
    free(this->done);
}



void parallel(const int GPU_index, const int dim, std::ofstream *times, const int wait, std::mutex* writeMutex) {
    spift* tester = new spift(dim, 16, 100000, GPU_index, wait);
    std::cout << "inited" << std::endl;
    //tester->generateShifts();
    //tester->iterate();
    std::thread shifts(&spift::generateShifts, tester);
    std::thread iter(&spift::iterate, tester);
    shifts.join();
    iter.join();
    writeMutex->lock();
    *times << dim << "\t" << wait << "\t" << tester->displayTime() << "\t" << GPU_index << std::endl;
    //tester->printResult();
    writeMutex->unlock();
    delete tester;

}



int main()
{
    /*
    std::ofstream times;
    times.open("timesGPU.txt");
    parallel(1, 4096, &times, 5);
    */
    auto writeMutex = new std::mutex();
    std::ofstream times;
    times.open("timesGPU.txt");
    for (int j = 0; j < 5; ++j) {
        for (int i = 0; i < 20; i++) {
            std::thread t0(parallel, 0, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t1(parallel, 1, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t2(parallel, 2, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t3(parallel, 3, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t4(parallel, 4, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t5(parallel, 5, pow(2, j + 9), &times, 5 * i, writeMutex);


            t0.join();
            t1.join();
            t2.join();
            t3.join();
            t4.join();
            t5.join();

            std::thread t00(parallel, 0, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t10(parallel, 1, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t20(parallel, 2, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t30(parallel, 3, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t40(parallel, 4, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t50(parallel, 5, pow(2, j + 9), &times, 5 * i, writeMutex);


            t00.join();
            t10.join();
            t20.join();
            t30.join();
            t40.join();
            t50.join();

            std::thread t000(parallel, 0, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t100(parallel, 1, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t200(parallel, 2, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t300(parallel, 3, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t400(parallel, 4, pow(2, j + 9), &times, 5 * i, writeMutex);
            std::thread t500(parallel, 5, pow(2, j + 9), &times, 5 * i, writeMutex);


            t000.join();
            t100.join();
            t200.join();
            t300.join();
            t400.join();
            t500.join();
        }
    }
    
    
    
    
    return 0;
}
